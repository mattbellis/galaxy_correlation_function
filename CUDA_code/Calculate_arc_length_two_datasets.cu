#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>

#include <hip/hip_runtime.h>
//#include <cutil_inline.h>

using namespace std;

#define SUBMATRIX_SIZE 10000
//#define NUM_BIN 5000
//#define HIST_MIN 0.0
//#define HIST_MAX 3.5 
#define NUM_BIN 27 // for log binning
//#define NUM_BIN 37 // for log binning
#define HIST_MIN 0.0 // for degrees
#define HIST_MAX 100.0 // for degrees

#define CONV_FACTOR 57.2957795 // 180/pi

//float bin_edges[30] = {0.001000,0.001585,0.002512,0.003981,0.006310,0.010000,0.010000,0.015849,0.025119,0.039811,0.063096,0.100000,0.100000,0.158489,0.251189,0.398107,0.630957,1.000000,1.000000,1.584893,2.511886,3.981072,6.309573,10.000000,10.000000,15.848932,25.118864,39.810717,63.095734,100.000000};

////////////////////////////////////////////////////////////////////////
__global__ void distance(float *a0, float *d0, float *a1, float *d1, int xind, int yind, int *dev_hist, float* dev_bin_edges, bool two_different_files=1)
{

    //float bin_edges[30] = {0.001000,0.001585,0.002512,0.003981,0.006310,0.010000,0.010000,0.015849,0.025119,0.039811,0.063096,0.100000,0.100000,0.158489,0.251189,0.398107,0.630957,1.000000,1.000000,1.584893,2.511886,3.981072,6.309573,10.000000,10.000000,15.848932,25.118864,39.810717,63.095734,100.000000};

    // For 27
    float bin_edges[NUM_BIN] = {0.0000,0.001000,0.001585,0.002512,0.003981,0.006310,0.010000,0.015849,0.025119,0.039811,0.063096,0.100000,0.158489,0.251189,0.398107,0.630957,1.000000,1.584893,2.511886,3.981072,6.309573,10.000000,15.848932,25.118864,39.810717,63.095734,100.000000};

    // For 37
    //float bin_edges[NUM_BIN] = {0.0000,0.001000,0.001389,0.001931,0.002683,0.003728,0.005179,0.007197,0.010000,0.013895,0.019307,0.026827,0.037276,0.051795,0.071969,0.100000,0.138950,0.193070,0.268270,0.372759,0.517947,0.719686,1.000000,1.389495,1.930698,2.682696,3.727594,5.179475,7.196857,10.000000,13.894955,19.306977,26.826958,37.275937,51.794747,71.968567,100.000000};


    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_idx = idx;
    idx += xind;

    float alpha = a0[idx], delta0 = d0[idx];
    float cos_d0 = cos(delta0), sin_d0 = sin(delta0), dist;

    int ymax = yind + SUBMATRIX_SIZE;
    int bin_index = 0; 
    int offset = 0;

    float a_diff, sin_a_diff, cos_a_diff;
    float cos_d1, sin_d1, numer, denom, mult1, mult2;    

    bool do_calc = 1;
    for(int i=yind; i<ymax; i++)
    {
        if (two_different_files)
        {
            do_calc = 1;
        }
        else // Doing the same file
        {
            if(idx > i)
                do_calc=1;
            else
                do_calc=0;
        }
        //if(idx > i) ///////// CHECK THIS
        if (do_calc)
        {
            a_diff = a1[i] - alpha;
            
            sin_a_diff = sin(a_diff);
            cos_a_diff = cos(a_diff);
  
            sin_d1 = sin(d1[i]);
            cos_d1 = cos(d1[i]);
 
            mult1 = cos_d1 * cos_d1 * sin_a_diff * sin_a_diff;
            mult2 = cos_d0 * sin_d1 - sin_d0 * cos_d1 * cos_a_diff;
            mult2 = mult2 * mult2;
           
            numer = sqrt(mult1 + mult2); 
       
            denom = sin_d0 *sin_d1 + cos_d0 * cos_d1 * cos_a_diff;
            
            //dist = atan(num);  
            dist = atan2(numer,denom);  
            dist *= CONV_FACTOR;  // Convert to degrees

            if(dist < HIST_MIN)
                bin_index = 0; 
            else if(dist >= HIST_MAX)
                bin_index = NUM_BIN + 1;
            else
                {
                    //bin_index = int(((dist - HIST_MIN) * NUM_BIN / HIST_MAX) +1);    
                    bin_index = 0;
                    for (int j=0;j<NUM_BIN-1;j++)
                    {
                        //bin_index = 5;
                        //if (dist>=0.1*j && dist<0.1*(j+1))
                        //if (dist>=dev_bin_edges[j] && dist<dev_bin_edges[j+1])
                        if (dist>=bin_edges[j] && dist<bin_edges[j+1])
                        {
                            bin_index = j+1;
                            break;
                        }
                    }
                }

            offset = ((NUM_BIN+2)*thread_idx);
            bin_index += offset;

           dev_hist[bin_index]++;

        }
    }
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    float *d_alpha0, *d_delta0;
    float *h_alpha0, *h_delta0;

    float *d_alpha1, *d_delta1;
    float *h_alpha1, *h_delta1;

    int NUM_PARTICLES;

    if (argc < 4)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    FILE *infile0, *infile1, *outfile ;
    infile0 = fopen(argv[1],"r");
    infile1 = fopen(argv[2],"r");
    outfile = fopen(argv[3], "w");

    bool two_different_files = 1;
    if (strcmp(argv[1],argv[2])==0)
    {
        two_different_files = 0;
        printf("Using the same file!\n");
    }

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    char axis_titles[256];
    char dummy[256];

    ////////////////////////////////////////////////////////////////////////////
    // Read in the first file
    ////////////////////////////////////////////////////////////////////////////
    
    fscanf(infile0, "%s %s %s", &axis_titles, &dummy, &axis_titles);
    fscanf(infile0, "%d", &NUM_PARTICLES);
   
    int size = NUM_PARTICLES * sizeof(float);    
    printf("SIZE0 # particles: %d\n",NUM_PARTICLES);

    h_alpha0 = (float*)malloc(size);
    h_delta0 = (float*)malloc(size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile0, "%f %s %f %s ", &h_alpha0[i], &dummy, &h_delta0[i], &dummy);
       //fscanf(infile, "%f%s %f ", &h_alpha[i], &dummy, &h_delta[i]);
       //printf("%e %s %e\n", h_alpha0[i], dummy, h_delta0[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    // Read in the second file
    ////////////////////////////////////////////////////////////////////////////
    
    fscanf(infile1, "%s %s %s", &axis_titles, &dummy, &axis_titles);
    fscanf(infile1, "%d", &NUM_PARTICLES);
    printf("SIZE1 # particles: %d\n",NUM_PARTICLES);
   
    h_alpha1 = (float*)malloc(size);
    h_delta1 = (float*)malloc(size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile1, "%f %s %f %s ", &h_alpha1[i], &dummy, &h_delta1[i], &dummy);
       //fscanf(infile, "%f%s %f ", &h_alpha[i], &dummy, &h_delta[i]);
       // printf("%e %s %e\n", h_alpha[i], dummy, h_delta[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    //allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;
    // Original
    //int size_hist = SUBMATRIX_SIZE * (NUM_BIN+2);
    //int size_hist_bytes = size_hist*sizeof(int);

    // Log binning
    //float h_bin_edges[30] = {0.001000,0.001585,0.002512,0.003981,0.006310,0.010000,0.010000,0.015849,0.025119,0.039811,0.063096,0.100000,0.100000,0.158489,0.251189,0.398107,0.630957,1.000000,1.000000,1.584893,2.511886,3.981072,6.309573,10.000000,10.000000,15.848932,25.118864,39.810717,63.095734,100.000000};

    // For 27 bins
    float h_bin_edges[NUM_BIN] = {0.0000,0.001000,0.001585,0.002512,0.003981,0.006310,0.010000,0.015849,0.025119,0.039811,0.063096,0.100000,0.158489,0.251189,0.398107,0.630957,1.000000,1.584893,2.511886,3.981072,6.309573,10.000000,15.848932,25.118864,39.810717,63.095734,100.000000};

    // For 37 bins
    //float h_bin_edges[NUM_BIN] = {0.0000,0.001000,0.001389,0.001931,0.002683,0.003728,0.005179,0.007197,0.010000,0.013895,0.019307,0.026827,0.037276,0.051795,0.071969,0.100000,0.138950,0.193070,0.268270,0.372759,0.517947,0.719686,1.000000,1.389495,1.930698,2.682696,3.727594,5.179475,7.196857,10.000000,13.894955,19.306977,26.826958,37.275937,51.794747,71.968567,100.000000};

    /*
    for (int i=0;i<NUM_BIN;i++)
    {
        printf("%d %f\n",i,h_bin_edges[i]);
    }
    printf("\n");
    */
    float *dev_bin_edges;
    hipMalloc((void **) &dev_bin_edges, (NUM_BIN*sizeof(float)));
    hipMemset(dev_bin_edges, 0, NUM_BIN);
    hipMemcpy(dev_bin_edges, h_bin_edges, NUM_BIN, hipMemcpyHostToDevice );
    //int nbins = 30;

    int size_hist = SUBMATRIX_SIZE * (NUM_BIN+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("size_hist: %d\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    unsigned long  *hist_array;

    hist_array =  (unsigned long*)malloc((NUM_BIN+2) * sizeof(unsigned long));
    memset(hist_array, 0, (NUM_BIN+2)*sizeof(unsigned long)); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    grid.x =100;
    block.x = SUBMATRIX_SIZE/grid.x; //NUM_PARTICLES/block.x;
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &d_alpha0, size );
    hipMalloc((void **) &d_delta0, size );

    hipMalloc((void **) &d_alpha1, size );
    hipMalloc((void **) &d_delta1, size );
    
    // Check to see if we allocated enough memory.
    if (0==d_alpha0 || 0==d_delta0 || 0==d_alpha1 || 0==d_delta1 || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(d_alpha0,0,size);
    hipMemset(d_delta0,0,size);
    hipMemset(d_alpha1,0,size);
    hipMemset(d_delta1,0,size);

    hipMemcpy(d_alpha0, h_alpha0, size, hipMemcpyHostToDevice );
    hipMemcpy(d_delta0, h_delta0, size, hipMemcpyHostToDevice );
    hipMemcpy(d_alpha1, h_alpha1, size, hipMemcpyHostToDevice );
    hipMemcpy(d_delta1, h_delta1, size, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices = NUM_PARTICLES / SUBMATRIX_SIZE;


    int bin_index = 0;
    for(int k = 0; k < num_submatrices; k++)
    {
        y = k*SUBMATRIX_SIZE;
//        printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices; j++)
        {
                x = j *SUBMATRIX_SIZE; 

                //printf("----\n");
                //printf("%d %d\t\t%d %d\n",k,y,j,x);
                //printf("----\n");

                hipMemset(dev_hist,0,size_hist_bytes);

                distance<<<grid,block>>>(d_alpha0, d_delta0,d_alpha1, d_delta1, x, y, dev_hist, dev_bin_edges, two_different_files);
                hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);


                for(int m=0; m<size_hist; m++)
                {

                    bin_index = m%(NUM_BIN+2);
                    //if(bin_index == 0)
                        //printf("\n");

                    //printf("%3i:%3i ", m, hist[m]);
                    //printf("%3i ", hist[m]);

                    hist_array[bin_index] += hist[m];
                }    
                //printf("\n");
        }  
    }

    unsigned long total = 0;
    float  bin_width = (HIST_MAX - HIST_MIN) / NUM_BIN;
    float bins_mid = 0;

    fprintf(outfile, "%s %s\n", "Angular Distance(radians)","Number of Entries");      
    for(int k=0; k<NUM_BIN+1; k++)
    {
       //bins_mid = bin_width*(k - 0.5);

       float lo = h_bin_edges[k];
       float hi = h_bin_edges[k+1];

       bins_mid = (hi+lo)/2.0;

       fprintf(outfile, "%.3e %s %lu \n", bins_mid, ",",  hist_array[k]);
       total += hist_array[k];

    }
    printf("total: %lu \n", total);
    
    fclose(infile0);
    fclose(infile1);
    fclose(outfile);

    free(h_alpha0);
    free(h_delta0);
    free(h_alpha1);
    free(h_delta1);
    free(hist);

    hipFree(d_alpha0);
    hipFree(d_delta0);  
    hipFree(d_alpha1);
    hipFree(d_delta1);  
    hipFree(dev_hist);
    hipFree(dev_bin_edges);

    return 0;
}  
//////////////////////////////////////////////////////////////////////
