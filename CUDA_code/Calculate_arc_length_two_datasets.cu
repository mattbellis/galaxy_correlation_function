#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
//#include <cutil_inline.h>

using namespace std;

#define SUBMATRIX_SIZE 10000
#define DEFAULT_NBINS 27 // for log binning

#define CONV_FACTOR 57.2957795 // 180/pi

////////////////////////////////////////////////////////////////////////
// Kernel to calculate angular distances between galaxies and histogram
// the distances.
////////////////////////////////////////////////////////////////////////
__global__ void distance(float *a0, float *d0, float *a1, float *d1, int xind, int yind, int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, bool log_binning=0, bool two_different_files=1)
{

    ////////////////////////////////////////////////////////////////////////////
    // Idx will keep track of which thread is being calculated within a given 
    // warp.
    ////////////////////////////////////////////////////////////////////////////
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // This should range to SUBMATRIX_SIZE

    int thread_idx = idx;
    idx += xind;

    int i=0;
    //int j=0;

    ////////////////////////////////////////////////////////////////////////////
    // Copy over a local version of the bin edges. This will prevent threads
    // from accessing the same memory that holds this information.
    ////////////////////////////////////////////////////////////////////////////
    /*
    float bin_edges[1024]; // This is hard-coded and will result in bugs if a histogram
    // with greater than 1024 bins is used.
    int bin_offset = thread_idx*nbins;
    for (i=0;i<nbins;i++)
    {
        bin_edges[i] = dev_bin_edges[i+bin_offset];
    }
    */

    //float hist_min = bin_edges[0];
    //float hist_max = bin_edges[nbins-1];

    float alpha = a0[idx], delta0 = d0[idx];
    float cos_d0 = cos(delta0), sin_d0 = sin(delta0), dist;

    int ymax = yind + SUBMATRIX_SIZE;
    int bin_index = 0; 
    int offset = 0;

    float a_diff, sin_a_diff, cos_a_diff;
    float cos_d1, sin_d1, numer, denom, mult1, mult2;    

    bool do_calc = 1;
    for(i=yind; i<ymax; i++)
    {
        if (two_different_files)
        {
            do_calc = 1;
        }
        else // Doing the same file
        {
            if(idx > i)
                do_calc=1;
            else
                do_calc=0;
        }
        //if(idx > i) ///////// CHECK THIS
        if (do_calc)
        {
            a_diff = a1[i] - alpha;

            sin_a_diff = sin(a_diff);
            cos_a_diff = cos(a_diff);

            sin_d1 = sin(d1[i]);
            cos_d1 = cos(d1[i]);

            mult1 = cos_d1 * cos_d1 * sin_a_diff * sin_a_diff;
            mult2 = cos_d0 * sin_d1 - sin_d0 * cos_d1 * cos_a_diff;
            mult2 = mult2 * mult2;

            numer = sqrt(mult1 + mult2); 

            denom = sin_d0 *sin_d1 + cos_d0 * cos_d1 * cos_a_diff;

            //dist = atan(num);  
            dist = atan2(numer,denom);  
            dist *= CONV_FACTOR;  // Convert to degrees

            if(dist < hist_min)
                bin_index = 0; 
            else if(dist >= hist_max)
                bin_index = nbins + 1;
            else
           if (!log_binning)
           {
               bin_index = int((dist-hist_min)/bin_width) + 1;
           }
           else // log binning
           {
               bin_index = int((log(dist)-log(hist_min))/bin_width) + 1;
           }

            /*
            {
                //bin_index = int(((dist - hist_min) * nbins / hist_max) +1);    
                bin_index = 0;
                for (j=0;j<nbins-1;j++)
                {
                    // This works
                    if (dist>=bin_edges[j] && dist<bin_edges[j+1])
                    {
                        bin_index = j+1;
                        break;
                    }
                }
            }
            */

            offset = ((nbins+2)*thread_idx);
            bin_index += offset;

            dev_hist[bin_index]++;

        }
    }
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Main
////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // Needed for parsing command-line arguments.
    extern char *optarg;
    extern int optind, optopt, opterr;
    int c;
    char *filename;
    char *outfilename = NULL;
    char defaultoutfilename[256];
    sprintf(defaultoutfilename,"default_out.dat");
    char *binning_filename = NULL;
    FILE *binning_file = NULL;

    float hist_lower_range = 0.0000001;
    float hist_upper_range = 0;
    int nbins = 100;
    float hist_bin_width = 0.05;
    bool log_binning_flag = 0; // False

    ////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////


    while ((c = getopt(argc, argv, "ab:o:L:N:lw:")) != -1) {
        switch(c) {
            case 'N':
                printf("N is set\n");
                nbins = atoi(optarg);
                break;
            case 'L':
                printf("L is set\n");
                hist_lower_range = atof(optarg);
                break;
            case 'w':
                hist_bin_width = atof(optarg);
                printf("Histogram bin width: %f\n",hist_bin_width);
                break;
            case 'l':
                log_binning_flag = 1;
                printf("Will use log binning.\n");
                break;
            case 'b':
                binning_filename = optarg;
                printf("Using binning information from file: %s\n",binning_filename);
                break;
            case 'o':
                outfilename = optarg;
                printf("Output filename is %s\n", outfilename);
                break;
            case '?':
                printf("unknown arg %c\n", optopt);
                break;
        }
    }

    if (argc < 2)
    {

        printf("\nMust pass in at least two input files on command line!\n");
        printf("\nUsage: ", argv[0] );
        //printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    // Set a default output file name, if none was passed in on the 
    // command line.
    if (outfilename == NULL) 
    {
        outfilename = defaultoutfilename;
        printf("Output filename is %s\n", outfilename);
    }

    float temp_lo = hist_lower_range;
    if (hist_upper_range == 0)
    {
        if (log_binning_flag==0)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = temp_lo + hist_bin_width;
                temp_lo = hist_upper_range;
            }
        }
        else
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = exp(log(temp_lo) + hist_bin_width);
                temp_lo = hist_upper_range;
            }
        }
    }
    printf("hist_upper_range: %f\n",hist_upper_range);

    //printf ("%d\n", optind);
    //printf ("%d\n", argc);
    //printf ("%d\n", optind);
    //printf ("%s\n", argv[optind]);
    //printf ("%s\n", argv[optind+1]);

    FILE *infile0, *infile1, *outfile ;
    infile0 = fopen(argv[optind],"r");
    infile1 = fopen(argv[optind+1],"r");
    //outfile = fopen(argv[3], "w");

    printf("Opening input file 0: %s\n",argv[optind]);
    printf("Opening input file 1: %s\n",argv[optind+1]);
    //printf("Outfilename: %s\n",outfilename);
    outfile = fopen(outfilename, "w");

    ////////////////////////////////////////////////////////////////////////////
    // Check to see if the two files are actually the same file.
    // This is the case for the DD and RR calculations and change slightly
    // the exact calculations being performed.
    ////////////////////////////////////////////////////////////////////////////
    bool two_different_files = 1;
    if (strcmp(argv[optind],argv[optind+1])==0)
    {
        two_different_files = 0;
        printf("Using the same file!\n");
    }
    printf("\n");
    ////////////////////////////////////////////////////////////////////////////
    // Now get the info from the device.
    ////////////////////////////////////////////////////////////////////////////
    printf("\n------ CUDA device diagnostics ------\n\n");

    int tot_gals = 40000;
    int nx = SUBMATRIX_SIZE;
    int ncalc = nx * nx;
    int gpu_mem_needed = int(tot_gals * sizeof(float)) * 5; // need to allocate gamma1, gamma2, ra, dec and output.
    printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", ncalc, gpu_mem_needed);

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
    }
    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");
    else
        printf("Found %d CUDA Capable device(s)\n", deviceCount);


    int dev, driverVersion = 0, runtimeVersion = 0;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);


        printf("  Warp size:                                     %d\n", deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
                deviceProp.maxThreadsDim[0],
                deviceProp.maxThreadsDim[1],
                deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
                deviceProp.maxGridSize[0],
                deviceProp.maxGridSize[1],
                deviceProp.maxGridSize[2]);

        // does this device have enough capcacity for the calculation?
        printf("\n*************\n");

        // check memory
        if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
        else
        {
            printf("Hurrah! This device has enough memory to perform this calculation\n");

            // check # threads

            int threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block.
            int blocksPerGrid = int(ceil(ncalc / threadsPerBlock)); // need nx*nx threads total
            if(deviceProp.maxThreadsDim[0] >blocksPerGrid) printf("FAILURE: Not enough threads on the device to do this calculation!\n");
            else
            {
                printf("Hurrah! This device supports enough threads to do this calculation\n");
                // how many kernels can we run at once on this machine?
                int n_mem = floor(deviceProp.totalGlobalMem / float(gpu_mem_needed));
                int n_threads = floor(threadsPerBlock * deviceProp.maxThreadsDim[0]*deviceProp.maxThreadsDim[1] / float(ncalc) ); // max # threads possible?

                printf("%d %d  \n",  n_threads, deviceProp.maxThreadsDim[0]);

                int max_kernels = 0;
                n_mem<n_threads ? max_kernels = n_mem : max_kernels = n_threads;

                printf(" you can run %d kernels at a time on this device without overloading the resources \n", max_kernels);
            }
        }

    }

    printf("\n------ End CUDA device diagnostics ------\n\n");
    ////////////////////////////////////////////////////////////////////////////

    float *d_alpha0, *d_delta0;
    float *h_alpha0, *h_delta0;

    float *d_alpha1, *d_delta1;
    float *h_alpha1, *h_delta1;

    //float *d_bin_edges;
    float *h_bin_edges;

    int NUM_GALAXIES;

    //////////////////////////////////////////////////////////////////////
    // Read in the file that defines the bin edges.
    ////////////////////////////////////////////////////////////////////////////

    /*
    int default_nbins = 27;
    float default_bin_edges[DEFAULT_NBINS] = {0.0000,0.001000,0.001585,0.002512,0.003981,0.006310,0.010000,0.015849,0.025119,0.039811,0.063096,0.100000,0.158489,0.251189,0.398107,0.630957,1.000000,1.584893,2.511886,3.981072,6.309573,10.000000,15.848932,25.118864,39.810717,63.095734,100.000000};

    int nbins=0;
    int size_bin_edges_array=0;
    float temp_bin_edges[4096];
    //FILE *binning_file = NULL;
    if (binning_filename != NULL)
    {
        printf("Binning filename: %s\n",binning_filename);
        binning_file = fopen(binning_filename,"r");

        while(fscanf(binning_file, "%f", &temp_bin_edges[nbins])!=EOF)
        {
            nbins++;
        }

        // Copy over the temp bin edges into the host array.
        // Note we are going to have one array of the bin edges for
        // *each thread*.
        size_bin_edges_array = nbins * sizeof(float)*SUBMATRIX_SIZE;
        h_bin_edges = (float*)malloc(size_bin_edges_array);
        printf("Size of host bin edges array: %d bytes\n",size_bin_edges_array);
        for (int i=0;i<SUBMATRIX_SIZE;i++)
        {
            int thread_index = i*nbins;
            for (int j=0;j<nbins;j++)
            {
                h_bin_edges[j+thread_index] = temp_bin_edges[j];
                //printf("h_bin_edges: %3d %f\n",j+thread_index,h_bin_edges[j+thread_index]);
            }
        }
    }
    else
    {
        // No file containing bin edges was passed in on the 
        // command line, so use the defaults.
        nbins = default_nbins;
        //size = nbins * sizeof(float);    
        //h_bin_edges = (float*)malloc(size);
        size_bin_edges_array = nbins * sizeof(float)*SUBMATRIX_SIZE;
        h_bin_edges = (float*)malloc(size_bin_edges_array);
        printf("Size of host bin edges array: %d bytes\n",size_bin_edges_array);
        for (int i=0;i<SUBMATRIX_SIZE;i++)
        {
            int thread_index = i*nbins;
            for (int j=0;j<nbins;j++)
            {
                h_bin_edges[j+thread_index] = default_bin_edges[j];
                //printf("h_bin_edges: %3d %f\n",j+thread_index,h_bin_edges[j+thread_index]);
            }
        }
        //for (int i=0;i<nbins;i++)
        //{
        //h_bin_edges[i] = default_bin_edges[i];
        ////printf("h_bin_edges: %3d %f\n",i,h_bin_edges[i]);
        //}
    }
    float hist_min = h_bin_edges[0];
    float hist_max = h_bin_edges[nbins-1];
    */

    ////////////////////////////////////////////////////////////////////////////
    // Finished defining the bin edges.
    ////////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////
    // Read in the galaxy files.
    ////////////////////////////////////////////////////////////////////////////
    // Read in the first file
    ////////////////////////////////////////////////////////////////////////////

    //fscanf(infile0, "%s %s %s", &axis_titles, &dummy, &axis_titles);
    fscanf(infile0, "%d", &NUM_GALAXIES);

    int size_of_galaxy_array = NUM_GALAXIES * sizeof(float);    
    printf("SIZE 0 # GALAXIES: %d\n",NUM_GALAXIES);

    h_alpha0 = (float*)malloc(size_of_galaxy_array);
    h_delta0 = (float*)malloc(size_of_galaxy_array);

    for(int i=0; i<NUM_GALAXIES; i++)
    {
        fscanf(infile0, "%f %f", &h_alpha0[i], &h_delta0[i]);
        //printf("%e %e\n", h_alpha0[i], h_delta0[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    // Read in the second file
    ////////////////////////////////////////////////////////////////////////////

    //fscanf(infile1, "%s %s %s", &axis_titles, &dummy, &axis_titles);
    fscanf(infile1, "%d", &NUM_GALAXIES);
    printf("SIZE 1 # GALAXIES: %d\n",NUM_GALAXIES);

    h_alpha1 = (float*)malloc(size_of_galaxy_array);
    h_delta1 = (float*)malloc(size_of_galaxy_array);

    for(int i=0; i<NUM_GALAXIES; i++)
    {
        fscanf(infile1, "%f %f", &h_alpha1[i], &h_delta1[i]);
        //printf("%e %e\n", h_alpha1[i], h_delta1[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    // Allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;
    //float *dev_bin_edges;

    //cudaMalloc((void **) &dev_bin_edges,size_bin_edges_array);
    //printf("Size of dev_bin_edges: %d bytes\n",size_bin_edges_array);
    //cudaMemset(dev_bin_edges, 0, nbins*SUBMATRIX_SIZE);
    //cudaMemcpy(dev_bin_edges, h_bin_edges,size_bin_edges_array, cudaMemcpyHostToDevice );

    int size_hist = SUBMATRIX_SIZE * (nbins+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("Size of histogram: %d bytes\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    unsigned long  *hist_array;

    int hist_array_size = (nbins+2) * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
    printf("Size of histogram array: %d bytes\n",hist_array_size);
    memset(hist_array,0,hist_array_size); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    grid.x =100; // Is this the number of blocks?
    block.x = SUBMATRIX_SIZE/grid.x; // Is this the number of threads per block? NUM_GALAXIES/block.x;
    // SUBMATRIX is the number of threads per warp? Per kernel call?
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &d_alpha0, size_of_galaxy_array );
    hipMalloc((void **) &d_delta0, size_of_galaxy_array );

    hipMalloc((void **) &d_alpha1, size_of_galaxy_array );
    hipMalloc((void **) &d_delta1, size_of_galaxy_array );

    // Check to see if we allocated enough memory.
    if (0==d_alpha0 || 0==d_delta0 || 0==d_alpha1 || 0==d_delta1 || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    // Initialize array to all 0's
    hipMemset(d_alpha0,0,size_of_galaxy_array);
    hipMemset(d_delta0,0,size_of_galaxy_array);
    hipMemset(d_alpha1,0,size_of_galaxy_array);
    hipMemset(d_delta1,0,size_of_galaxy_array);

    hipMemcpy(d_alpha0, h_alpha0, size_of_galaxy_array, hipMemcpyHostToDevice );
    hipMemcpy(d_delta0, h_delta0, size_of_galaxy_array, hipMemcpyHostToDevice );
    hipMemcpy(d_alpha1, h_alpha1, size_of_galaxy_array, hipMemcpyHostToDevice );
    hipMemcpy(d_delta1, h_delta1, size_of_galaxy_array, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices = NUM_GALAXIES / SUBMATRIX_SIZE;

    printf("Breaking down the calculations.\n");
    printf("Number of submatrices: %dx%d\n",num_submatrices,num_submatrices);
    printf("Number of calculations per submatrices: %dx%d\n",SUBMATRIX_SIZE,SUBMATRIX_SIZE);


    int bin_index = 0;
    for(int k = 0; k < num_submatrices; k++)
    {
        y = k*SUBMATRIX_SIZE;
        //printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices; j++)
        {
            x = j *SUBMATRIX_SIZE; 

            //printf("----\n");
            //printf("%d %d\t\t%d %d\n",k,y,j,x);
            //printf("----\n");

            // Set the histogram to all zeros each time.
            hipMemset(dev_hist,0,size_hist_bytes);

//__global__ void distance(float *a0, float *d0, float *a1, float *d1, int xind, int yind, int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, bool log_binning=0, bool two_different_files=1)
            distance<<<grid,block>>>(d_alpha0, d_delta0,d_alpha1, d_delta1, x, y, dev_hist, hist_lower_range, hist_upper_range, nbins, hist_bin_width, log_binning_flag, two_different_files);
            hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);

            ////////////////////////////////////////////////////////////////////
            // Sum up the histograms from each thread (hist).
            ////////////////////////////////////////////////////////////////////
            for(int m=0; m<size_hist; m++)
            {
                bin_index = m%(nbins+2);
                hist_array[bin_index] += hist[m];
            }    
        }  
    }

    unsigned long total = 0;
    //float  bin_width = (hist_upper_range - hist_lower_range) / nbins;
    float bins_mid = 0;

    fprintf(outfile, "%s %s\n", "Angular Distance(radians)","Number of Entries");      
    float lo = hist_lower_range;
    float hi = 0;
    //printf("hist_lower_range: %f\n",hist_lower_range);
    //printf("hist_upper_range: %f\n",hist_upper_range);
    //printf("hist_bin_width: %f\n",hist_bin_width);
    for(int k=0; k<nbins+1; k++)
    {
        //bins_mid = bin_width*(k - 0.5);

        //float lo = h_bin_edges[k];
        //float hi = h_bin_edges[k+1];
        if (!log_binning_flag)
        {
            hi = lo + hist_bin_width;
        }
        else
        {
            //printf("lo: %f\t\tlog(lo): %f\n",lo,log(lo));
            hi = exp(log(lo) + hist_bin_width);
        }

        bins_mid = (hi+lo)/2.0;

        fprintf(outfile, "%.3e %s %lu \n", bins_mid, ",",  hist_array[k]);
        total += hist_array[k];

        lo = hi;

    }
    printf("total: %lu \n", total);

    fclose(infile0);
    fclose(infile1);
    fclose(outfile);

    free(h_alpha0);
    free(h_delta0);
    free(h_alpha1);
    free(h_delta1);
    free(hist);

    hipFree(d_alpha0);
    hipFree(d_delta0);  
    hipFree(d_alpha1);
    hipFree(d_delta1);  
    hipFree(dev_hist);
    //cudaFree(dev_bin_edges);

    return 0;
}  
//////////////////////////////////////////////////////////////////////
