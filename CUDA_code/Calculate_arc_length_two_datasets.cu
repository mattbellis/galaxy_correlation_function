#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
//#include <cutil_inline.h>

using namespace std;

#define SUBMATRIX_SIZE 10000
#define DEFAULT_NBINS 27 // for log binning

#define CONV_FACTOR 57.2957795 // 180/pi

////////////////////////////////////////////////////////////////////////
// Kernel to calculate angular distances between galaxies and histogram
// the distances.
////////////////////////////////////////////////////////////////////////
__global__ void distance(float *a0, float *d0, float *a1, float *d1, int xind, int yind, int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, int log_binning=0, bool two_different_files=1, float conv_factor_angle=57.2957795)
{

    ////////////////////////////////////////////////////////////////////////////
    // Idx will keep track of which thread is being calculated within a given 
    // warp.
    ////////////////////////////////////////////////////////////////////////////
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // This should range to SUBMATRIX_SIZE

    int thread_idx = idx;
    idx += xind;

    int i=0;

    float alpha_rad = a0[idx];
    float delta0 = d0[idx];
    float cos_d0 = cos(delta0);
    float sin_d0 = sin(delta0);
    float dist;

    int ymax = yind + SUBMATRIX_SIZE;
    int bin_index = 0; 
    int offset = 0;

    float a_diff, sin_a_diff, cos_a_diff;
    float cos_d1, sin_d1, numer, denom, mult1, mult2;    
    float d1_rad;

    bool do_calc = 1;
    for(i=yind; i<ymax; i++)
    {
        if (two_different_files)
        {
            do_calc = 1;
        }
        else // Doing the same file
        {
            if(idx > i)
                do_calc=1;
            else
                do_calc=0;
        }
        //if(idx > i) ///////// CHECK THIS
        if (do_calc)
        {
            a_diff = a1[i] - alpha_rad;
            d1_rad = d1[i];

            sin_a_diff = sin(a_diff);
            cos_a_diff = cos(a_diff);

            sin_d1 = sin(d1_rad);
            cos_d1 = cos(d1_rad);

            mult1 = cos_d1 * cos_d1 * sin_a_diff * sin_a_diff;
            mult2 = cos_d0 * sin_d1 - sin_d0 * cos_d1 * cos_a_diff;
            mult2 = mult2 * mult2;

            numer = sqrt(mult1 + mult2); 

            denom = sin_d0 *sin_d1 + cos_d0 * cos_d1 * cos_a_diff;

            //dist = atan(num);  
            dist = atan2(numer,denom);  
            dist *= conv_factor_angle;  // Convert to degrees or what have you.

            if(dist < hist_min)
                bin_index = 0; 
            else if(dist >= hist_max)
                bin_index = nbins + 1;
            else
           if (log_binning==0)
           {
               bin_index = int((dist-hist_min)/bin_width) + 1;
           }
            else if (log_binning==1)// log binning
           {
               bin_index = int((log(dist)-log(hist_min))/bin_width) + 1;
           }
            else if (log_binning==2)// log 10 binning
           {
               bin_index = int((log10(dist)-log10(hist_min))/bin_width) + 1;
           }

            offset = ((nbins+2)*thread_idx);
            bin_index += offset;

            dev_hist[bin_index]++;

        }
    }
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Main
////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // Needed for parsing command-line arguments.
    extern char *optarg;
    extern int optind, optopt, opterr;
    int c;
    char *filename;
    char *outfilename = NULL;
    char defaultoutfilename[256];
    sprintf(defaultoutfilename,"default_out.dat");
    char *binning_filename = NULL;
    FILE *binning_file = NULL;

    float hist_lower_range = 0.0000001;
    float hist_upper_range = 0;
    int nbins = 100;
    float hist_bin_width = 0.05;
    int log_binning_flag = 0; // False

    float scale_factor = 1.0; // For if we need to convert input to arcsec or arcmin
    float conv_factor_angle = 57.2957795; // 180/pi // For if we need to convert arcdistance to arcsec or arcmin

    ////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////


    while ((c = getopt(argc, argv, "ab:o:L:N:l:w:sm")) != -1) {
        switch(c) {
            case 'N':
                printf("N is set\n");
                nbins = atoi(optarg);
                break;
            case 'L':
                printf("L is set\n");
                hist_lower_range = atof(optarg);
                break;
            case 'w':
                hist_bin_width = atof(optarg);
                printf("Histogram bin width: %f\n",hist_bin_width);
                break;
            case 'l':
                log_binning_flag = atoi(optarg);
                printf("Will use log binning.\n");
                break;
            case 'b':
                binning_filename = optarg;
                printf("Using binning information from file: %s\n",binning_filename);
                break;
            case 's':
                scale_factor = 206264.0; // To convert arcseconds to radians.
                conv_factor_angle *= 3600.0; // convert radians to arcseconds.
                printf("Reading in values assuming they are arcseconds.\n");
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                break;
            case 'm':
                scale_factor = 3437.74677; // To convert arcminutes to radians.
                conv_factor_angle *= 60.0; // convert radians to arcminutes.
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                printf("Reading in values assuming they are arcminutes.\n");
                break;
            case 'o':
                outfilename = optarg;
                printf("Output filename is %s\n", outfilename);
                break;
            case '?':
                printf("unknown arg %c\n", optopt);
                break;
        }
    }

    if (argc < 2)
    {

        printf("\nMust pass in at least two input files on command line!\n");
        printf("\nUsage: ", argv[0] );
        //printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    // Set a default output file name, if none was passed in on the 
    // command line.
    if (outfilename == NULL) 
    {
        outfilename = defaultoutfilename;
        printf("Output filename is %s\n", outfilename);
    }

    float temp_lo = hist_lower_range;
    if (hist_upper_range == 0)
    {
        if (log_binning_flag==0)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = temp_lo + hist_bin_width;
                temp_lo = hist_upper_range;
            }
        }
        else if (log_binning_flag==1)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = exp(log(temp_lo) + hist_bin_width);
                temp_lo = hist_upper_range;
            }
        }
        else if (log_binning_flag==2)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = pow(10,(log10(temp_lo) + hist_bin_width));
                temp_lo = hist_upper_range;
            }
        }
    }
    printf("hist_upper_range: %f\n",hist_upper_range);

    FILE *infile0, *infile1, *outfile ;
    infile0 = fopen(argv[optind],"r");
    infile1 = fopen(argv[optind+1],"r");

    printf("Opening input file 0: %s\n",argv[optind]);
    printf("Opening input file 1: %s\n",argv[optind+1]);
    outfile = fopen(outfilename, "w");

    ////////////////////////////////////////////////////////////////////////////
    // Check to see if the two files are actually the same file.
    // This is the case for the DD and RR calculations and change slightly
    // the exact calculations being performed.
    ////////////////////////////////////////////////////////////////////////////
    bool two_different_files = 1;
    if (strcmp(argv[optind],argv[optind+1])==0)
    {
        two_different_files = 0;
        printf("Using the same file!\n");
    }
    printf("\n");
    ////////////////////////////////////////////////////////////////////////////
    // Now get the info from the device.
    ////////////////////////////////////////////////////////////////////////////
    printf("\n------ CUDA device diagnostics ------\n\n");

    int tot_gals = 40000;
    int nx = SUBMATRIX_SIZE;
    int ncalc = nx * nx;
    int gpu_mem_needed = int(tot_gals * sizeof(float)) * 5; // need to allocate gamma1, gamma2, ra, dec and output.
    printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", ncalc, gpu_mem_needed);

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
    }
    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");
    else
        printf("Found %d CUDA Capable device(s)\n", deviceCount);


    int dev, driverVersion = 0, runtimeVersion = 0;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);


        printf("  Warp size:                                     %d\n", deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
                deviceProp.maxThreadsDim[0],
                deviceProp.maxThreadsDim[1],
                deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
                deviceProp.maxGridSize[0],
                deviceProp.maxGridSize[1],
                deviceProp.maxGridSize[2]);

        // does this device have enough capcacity for the calculation?
        printf("\n*************\n");

        // check memory
        if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
        else
        {
            printf("Hurrah! This device has enough memory to perform this calculation\n");

            // check # threads

            int threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block.
            int blocksPerGrid = int(ceil(ncalc / threadsPerBlock)); // need nx*nx threads total
            if(deviceProp.maxThreadsDim[0] >blocksPerGrid) printf("FAILURE: Not enough threads on the device to do this calculation!\n");
            else
            {
                printf("Hurrah! This device supports enough threads to do this calculation\n");
                // how many kernels can we run at once on this machine?
                int n_mem = floor(deviceProp.totalGlobalMem / float(gpu_mem_needed));
                int n_threads = floor(threadsPerBlock * deviceProp.maxThreadsDim[0]*deviceProp.maxThreadsDim[1] / float(ncalc) ); // max # threads possible?

                printf("%d %d  \n",  n_threads, deviceProp.maxThreadsDim[0]);

                int max_kernels = 0;
                n_mem<n_threads ? max_kernels = n_mem : max_kernels = n_threads;

                printf(" you can run %d kernels at a time on this device without overloading the resources \n", max_kernels);
            }
        }

    }

    printf("\n------ End CUDA device diagnostics ------\n\n");
    ////////////////////////////////////////////////////////////////////////////

    float *d_alpha0, *d_delta0;
    float *h_alpha0, *h_delta0;

    float *d_alpha1, *d_delta1;
    float *h_alpha1, *h_delta1;

    //float *d_bin_edges;
    float *h_bin_edges;

    int NUM_GALAXIES;

    //////////////////////////////////////////////////////////////////////
    // Read in the galaxy files.
    ////////////////////////////////////////////////////////////////////////////
    // Read in the first file
    ////////////////////////////////////////////////////////////////////////////

    //fscanf(infile0, "%s %s %s", &axis_titles, &dummy, &axis_titles);
    fscanf(infile0, "%d", &NUM_GALAXIES);

    int size_of_galaxy_array = NUM_GALAXIES * sizeof(float);    
    printf("SIZE 0 # GALAXIES: %d\n",NUM_GALAXIES);

    h_alpha0 = (float*)malloc(size_of_galaxy_array);
    h_delta0 = (float*)malloc(size_of_galaxy_array);
    float temp0, temp1;

    for(int i=0; i<NUM_GALAXIES; i++)
    {
        fscanf(infile0, "%f %f", &temp0, &temp1);
        h_alpha0[i] = temp0/scale_factor;
        h_delta0[i] = temp1/scale_factor;
        //fscanf(infile0, "%f %f", &h_alpha0[i]*scale_factor, &h_delta0[i]*scale_factor);
        //printf("%e %e\n", h_alpha0[i], h_delta0[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    // Read in the second file
    ////////////////////////////////////////////////////////////////////////////

    //fscanf(infile1, "%s %s %s", &axis_titles, &dummy, &axis_titles);
    fscanf(infile1, "%d", &NUM_GALAXIES);
    printf("SIZE 1 # GALAXIES: %d\n",NUM_GALAXIES);

    h_alpha1 = (float*)malloc(size_of_galaxy_array);
    h_delta1 = (float*)malloc(size_of_galaxy_array);

    for(int i=0; i<NUM_GALAXIES; i++)
    {
        fscanf(infile0, "%f %f", &temp0, &temp1);
        h_alpha1[i] = temp0/scale_factor;
        h_delta1[i] = temp1/scale_factor;
        //fscanf(infile1, "%f %f", &h_alpha1[i]*scale_factor, &h_delta1[i]*scale_factor);
        //printf("%e %e\n", h_alpha1[i], h_delta1[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    // Allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;

    int size_hist = SUBMATRIX_SIZE * (nbins+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("Size of histogram: %d bytes\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    unsigned long  *hist_array;

    int hist_array_size = (nbins+2) * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
    printf("Size of histogram array: %d bytes\n",hist_array_size);
    memset(hist_array,0,hist_array_size); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    grid.x =100; // Is this the number of blocks?
    block.x = SUBMATRIX_SIZE/grid.x; // Is this the number of threads per block? NUM_GALAXIES/block.x;
    // SUBMATRIX is the number of threads per warp? Per kernel call?
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &d_alpha0, size_of_galaxy_array );
    hipMalloc((void **) &d_delta0, size_of_galaxy_array );

    hipMalloc((void **) &d_alpha1, size_of_galaxy_array );
    hipMalloc((void **) &d_delta1, size_of_galaxy_array );

    // Check to see if we allocated enough memory.
    if (0==d_alpha0 || 0==d_delta0 || 0==d_alpha1 || 0==d_delta1 || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    // Initialize array to all 0's
    hipMemset(d_alpha0,0,size_of_galaxy_array);
    hipMemset(d_delta0,0,size_of_galaxy_array);
    hipMemset(d_alpha1,0,size_of_galaxy_array);
    hipMemset(d_delta1,0,size_of_galaxy_array);

    hipMemcpy(d_alpha0, h_alpha0, size_of_galaxy_array, hipMemcpyHostToDevice );
    hipMemcpy(d_delta0, h_delta0, size_of_galaxy_array, hipMemcpyHostToDevice );
    hipMemcpy(d_alpha1, h_alpha1, size_of_galaxy_array, hipMemcpyHostToDevice );
    hipMemcpy(d_delta1, h_delta1, size_of_galaxy_array, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices = NUM_GALAXIES / SUBMATRIX_SIZE;

    printf("Breaking down the calculations.\n");
    printf("Number of submatrices: %dx%d\n",num_submatrices,num_submatrices);
    printf("Number of calculations per submatrices: %dx%d\n",SUBMATRIX_SIZE,SUBMATRIX_SIZE);


    int bin_index = 0;
    for(int k = 0; k < num_submatrices; k++)
    {
        y = k*SUBMATRIX_SIZE;
        //printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices; j++)
        {
            x = j *SUBMATRIX_SIZE; 

            //printf("----\n");
            //printf("%d %d\t\t%d %d\n",k,y,j,x);
            //printf("----\n");

            // Set the histogram to all zeros each time.
            hipMemset(dev_hist,0,size_hist_bytes);

//__global__ void distance(float *a0, float *d0, float *a1, float *d1, int xind, int yind, int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, bool log_binning=0, bool two_different_files=1)
            distance<<<grid,block>>>(d_alpha0, d_delta0,d_alpha1, d_delta1, x, y, dev_hist, hist_lower_range, hist_upper_range, nbins, hist_bin_width, log_binning_flag, two_different_files,conv_factor_angle);
            hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);

            ////////////////////////////////////////////////////////////////////
            // Sum up the histograms from each thread (hist).
            ////////////////////////////////////////////////////////////////////
            for(int m=0; m<size_hist; m++)
            {
                bin_index = m%(nbins+2);
                hist_array[bin_index] += hist[m];
            }    
        }  
    }

    unsigned long total = 0;
    //float  bin_width = (hist_upper_range - hist_lower_range) / nbins;
    float bins_mid = 0;

    fprintf(outfile, "%s %s\n", "Angular Distance(radians)","Number of Entries");      
    float lo = hist_lower_range;
    float hi = 0;
    //printf("hist_lower_range: %f\n",hist_lower_range);
    //printf("hist_upper_range: %f\n",hist_upper_range);
    //printf("hist_bin_width: %f\n",hist_bin_width);
    for(int k=0; k<nbins+1; k++)
    {
        //bins_mid = bin_width*(k - 0.5);

        //float lo = h_bin_edges[k];
        //float hi = h_bin_edges[k+1];
        if (log_binning_flag==0)
        {
            hi = lo + hist_bin_width;
        }
        else if (log_binning_flag==1)
        {
            //printf("lo: %f\t\tlog(lo): %f\n",lo,log(lo));
            hi = exp(log(lo) + hist_bin_width);
        }
        else if (log_binning_flag==2)
        {
            //printf("lo: %f\t\tlog10(lo): %f\n",lo,log10(lo));
            hi = pow(10,(log10(lo) + hist_bin_width));
        }

        bins_mid = (hi+lo)/2.0;

        fprintf(outfile, "%.3e %s %lu \n", bins_mid, ",",  hist_array[k]);
        total += hist_array[k];

        lo = hi;

    }
    printf("total: %lu \n", total);

    fclose(infile0);
    fclose(infile1);
    fclose(outfile);

    free(h_alpha0);
    free(h_delta0);
    free(h_alpha1);
    free(h_delta1);
    free(hist);

    hipFree(d_alpha0);
    hipFree(d_delta0);  
    hipFree(d_alpha1);
    hipFree(d_delta1);  
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////
